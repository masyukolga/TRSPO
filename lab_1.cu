#include <stdio.h>
#include <hip/hip_runtime.h>

#define N 	256
#define 	THREADS_PER_BLOCK 256
#define BLOCKS 	ceil( float(N) / THREADS_PER_BLOCK )

__global__ void vector_addition (double *a, double *b, double *result)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if(idx < N) 
	result[idx] = a[idx] + b[idx];
}

int main()
{
   
    double *a = new double[N];
    double *b = new double[N];
    double *res = new double[N];

    double *d_a, *d_b, *d_res;
    hipMalloc(&d_a, N*sizeof(double));
    hipMalloc(&d_b, N*sizeof(double));
    hipMalloc(&d_res, N*sizeof(double));

    for(int i=0; i<N; i++)
    {
        a[i] = rand()%10;
        b[i] = rand()%10;
    }

    hipMemcpy(d_a, a, N*sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N*sizeof(double), hipMemcpyHostToDevice);

    vector_addition<<< BLOCKS, THREADS_PER_BLOCK >>>(d_a, d_b, d_res);

    hipMemcpy(res, d_res, N*sizeof(double), hipMemcpyDeviceToHost);

    for(int i = 0; i < N; i++){
	printf("%lf\n", res[i]);
    }

    delete a;
    delete b;
    delete res;

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_res);


    return 0;
}
